#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ // declares kernels, variables declared here are device variables
void saxpy(int n, float a, float *x, float *y) 
// n, a, i will be stored in registers by each thread
// *x and *y must be pointers to device memory space
{
  // blockDim = dimensions of each block (256)
  // threadIdx - index of thread, blockIdx - index of block
  // i becomes a global index that can be used to access array elements 
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  
  printf("%i \n" , N);
  
  // Host Code
  float *x, *y, *d_x, *d_y; 
  
  // x & y are host arrays 
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  // D_x & d_y point to device arrays, allocated with hipMalloc
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  // Initialize x to array of ones. y to arrays of twos
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // destination, source, size, direction of copy
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  // triple chevrons define execution configuration
  // <<<grid of thread blocks, threads>>> 
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  // Copy back to host after kernel has been run
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %fn", maxError);
}
